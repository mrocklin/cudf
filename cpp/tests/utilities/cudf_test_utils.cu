/*
 * Copyright (c) 2018, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "cudf_test_utils.cuh"

namespace {

struct column_printer {
  template <typename ColumnType>
  void operator()(gdf_column const* the_column) {

    gdf_size_type const num_rows{the_column->size};

    ColumnType const* col_data{
        static_cast<ColumnType const*>(the_column->data)};

    std::vector<ColumnType> h_data(num_rows);
    hipMemcpy(h_data.data(), col_data, num_rows * sizeof(ColumnType),
               hipMemcpyDeviceToHost);

    std::vector<gdf_valid_type> h_mask(gdf_valid_allocation_size(num_rows), ~gdf_valid_type{0});
    if (nullptr != the_column->valid) {
      hipMemcpy(h_mask.data(), the_column->valid,
                 gdf_num_bitmask_elements(num_rows) * sizeof(gdf_valid_type), hipMemcpyDeviceToHost);
    }

    for (gdf_size_type i = 0; i < num_rows; ++i) {
      // If the element is valid, print it's value
      if (true == gdf_is_valid(h_mask.data(), i)) {
        std::cout << h_data[i] << " ";
      }
      // Otherwise, print an @ to represent a null value
      else {
        std::cout << "@"
                  << " ";
      }
    }
    std::cout << std::endl;
  }
};
}

void print_gdf_column(gdf_column const * the_column)
{
    cudf::type_dispatcher(the_column->dtype, column_printer{}, the_column);
}

void print_valid_data(const gdf_valid_type *validity_mask, 
                      const size_t num_rows)
{
  hipError_t error;
  hipPointerAttribute_t attrib;
  hipPointerGetAttributes(&attrib, validity_mask);
  error = hipGetLastError();

  std::vector<gdf_valid_type> h_mask(gdf_valid_allocation_size(num_rows));
  if (error != hipErrorInvalidValue && attrib.memoryType == hipMemoryTypeDevice)
    hipMemcpy(h_mask.data(), validity_mask, gdf_valid_allocation_size(num_rows), hipMemcpyDeviceToHost);
  else
    memcpy(h_mask.data(), validity_mask, gdf_valid_allocation_size(num_rows));

  std::transform(
      h_mask.begin(), h_mask.begin() + gdf_num_bitmask_elements(num_rows),
      std::ostream_iterator<std::string>(std::cout, " "), [](gdf_valid_type x) {
        auto bits = std::bitset<GDF_VALID_BITSIZE>(x).to_string('@');
        return std::string(bits.rbegin(), bits.rend());
      });
  std::cout << std::endl;
}

