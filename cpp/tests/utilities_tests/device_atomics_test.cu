#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <tests/utilities/column_wrapper.cuh>
#include <tests/utilities/cudf_test_fixtures.h>
#include <utilities/wrapper_types.hpp>
#include <utilities/device_atomics.cuh>

#include <gmock/gmock.h>
#include <gtest/gtest.h>

#include <thrust/device_vector.h>
#include <thrust/transform.h>

#include <bitset>
#include <cstdint>
#include <random>

template<typename T>
__global__
void gpu_atomic_test(T *result, T *data, size_t size)
{
    size_t id   = blockIdx.x * blockDim.x + threadIdx.x;
    size_t step = blockDim.x * gridDim.x;

    for (; id < size; id += step) {
        atomicAdd(&result[0], data[id]);
        atomicMin(&result[1], data[id]);
        atomicMax(&result[2], data[id]);
        atomicAdd(&result[3], data[id]);
    }
}

template<typename T>
__global__
void gpu_atomicCAS_test(T *result, T *data, size_t size)
{
    size_t id   = blockIdx.x * blockDim.x + threadIdx.x;
    size_t step = blockDim.x * gridDim.x;

    for (; id < size; id += step) {
        T* addr = &result[0];
        T update_value = data[id];

        T old_value = *addr;
        T assumed;

        do {
            assumed  = old_value;
            const T new_value = old_value + update_value;

            old_value = atomicCAS(addr, assumed, new_value);
        } while (assumed != old_value);
    }
}

// TODO: remove these explicit instantiation for kernels
// At TYPED_TEST, the kernel for TypeParam of `wrapper` types won't be instantiated,
// then kenrel call failed by `hipErrorInvalidDeviceFunction`

template  __global__ void gpu_atomic_test<cudf::date32>(cudf::date32 *result, cudf::date32 *data, size_t size);
template  __global__ void gpu_atomic_test<cudf::date64>(cudf::date64 *result, cudf::date64 *data, size_t size);
template  __global__ void gpu_atomic_test<cudf::category>(cudf::category *result, cudf::category *data, size_t size);
template  __global__ void gpu_atomic_test<cudf::timestamp>(cudf::timestamp *result, cudf::timestamp *data, size_t size);

template  __global__ void gpu_atomicCAS_test<cudf::date32>(cudf::date32 *result, cudf::date32 *data, size_t size);
template  __global__ void gpu_atomicCAS_test<cudf::date64>(cudf::date64 *result, cudf::date64 *data, size_t size);
template  __global__ void gpu_atomicCAS_test<cudf::category>(cudf::category *result, cudf::category *data, size_t size);
template  __global__ void gpu_atomicCAS_test<cudf::timestamp>(cudf::timestamp *result, cudf::timestamp *data, size_t size);

// ---------------------------------------------

template <typename T>
struct AtomicsTest : public GdfTest {
};

using TestingTypes = ::testing::Types<
    int8_t, int16_t, int32_t, int64_t, float, double,
    cudf::date32, cudf::date64, cudf::timestamp, cudf::category
    >;

TYPED_TEST_CASE(AtomicsTest, TestingTypes);

// tests for atomicAdd/Min/Max
TYPED_TEST(AtomicsTest, atomicOps)
{
    using T = TypeParam;
    std::vector<int> v({6, -14, 13, 64, -13, -20, 45});
    int exact[3];
    exact[0] = std::accumulate(v.begin(), v.end(), 0);
    exact[1] = *( std::min_element(v.begin(), v.end()) );
    exact[2] = *( std::max_element(v.begin(), v.end()) );
    size_t vec_size = v.size();

    // std::vector<T> v_type({6, -14, 13, 64, -13, -20, 45}));
    // use transform from std::vector<int> instead.
    std::vector<T> v_type(vec_size);
    std::transform(v.begin(), v.end(), v_type.begin(),
        [](int x) { T t(x) ; return t; } );

    std::vector<T> result_init(4);
    result_init[0] = T{0};
    result_init[1] = std::numeric_limits<T>::max();
    result_init[2] = std::numeric_limits<T>::min();
    result_init[3] = T{0};

    thrust::device_vector<T> dev_result(result_init);
    thrust::device_vector<T> dev_data(v_type);

    hipDeviceSynchronize();
    CUDA_CHECK_LAST();

    gpu_atomic_test<T> <<<1, vec_size>>> (
        reinterpret_cast<T*>( dev_result.data().get() ),
        reinterpret_cast<T*>( dev_data.data().get() ),
        vec_size);

    hipDeviceSynchronize();
    CUDA_CHECK_LAST();

    thrust::host_vector<T> host_result(dev_result);
    hipDeviceSynchronize();

    CUDA_CHECK_LAST();

    EXPECT_EQ(host_result[0], T(exact[0])) << "atomicAdd test failed";
    EXPECT_EQ(host_result[1], T(exact[1])) << "atomicMin test failed";
    EXPECT_EQ(host_result[2], T(exact[2])) << "atomicMax test failed";
    EXPECT_EQ(host_result[3], T(exact[0])) << "atomicAdd test(2) failed";
}

// ------------------------------------------------------------------------------------------------

template <typename T>
struct AtomicsCASTest : public GdfTest {
};

// TODO: add `int8_t`, `int16_t` if `atomicCAS` supports
using TestingTypesForCAS = ::testing::Types<
    int32_t, int64_t, float, double,
    cudf::date32, cudf::date64, cudf::timestamp, cudf::category
    >;

TYPED_TEST_CASE(AtomicsCASTest, TestingTypesForCAS);

// tests for atomicCAS
TYPED_TEST(AtomicsCASTest, atomicCAS)
{
    using T = TypeParam;
    std::vector<int> v({6, -14, 13, 64, -13, -20, 45});
    int exact = std::accumulate(v.begin(), v.end(), 0);
    size_t vec_size = v.size();

    // std::vector<T> v_type({6, -14, 13, 64, -13, -20, 45}));
    // use transform from std::vector<int> instead.
    std::vector<T> v_type(vec_size);
    std::transform(v.begin(), v.end(), v_type.begin(),
        [](int x) { T t(x) ; return t; } );

    std::vector<T> result_init({T{0}});

    thrust::device_vector<T> dev_result(result_init);
    thrust::device_vector<T> dev_data(v_type);

    hipDeviceSynchronize();
    CUDA_CHECK_LAST();

    gpu_atomicCAS_test<T> <<<1, vec_size>>> (
        reinterpret_cast<T*>( dev_result.data().get() ),
        reinterpret_cast<T*>( dev_data.data().get() ),
        vec_size);

    hipDeviceSynchronize();
    CUDA_CHECK_LAST();

    thrust::host_vector<T> host_result(dev_result);
    hipDeviceSynchronize();

    CUDA_CHECK_LAST();

    EXPECT_EQ(host_result[0], T(exact)) << "atomicCAS test failed";
}


