#include "hip/hip_runtime.h"
#include "cudf.h"
#include "rmm/rmm.h"
#include "utilities/cudf_utils.h"
#include "utilities/error_utils.hpp"
#include "bitmask/bit_mask.cuh"
#include "utilities/type_dispatcher.hpp"
#include "bitmask/legacy_bitmask.hpp"

#include <cub/device/device_scan.cuh>

namespace { //anonymous

#define COPYMASK_BLOCK_SIZE 1024

    template <class T>
    __global__
        void gpu_copy_and_replace_nulls(
            const T *data, const gdf_valid_type *mask,
            gdf_size_type size, T *results, T identity)
    {
        gdf_size_type id = threadIdx.x + blockIdx.x * blockDim.x;

        while (id < size) {
            results[id] = (gdf_is_valid(mask, id)) ? data[id] : identity;
            id += blockDim.x * gridDim.x;
        }
    }

/* --------------------------------------------------------------------------*/
/**
 * @brief Copy data stream and replace nulls by a scholar value
 *
 * @Param[in] data The stream to be copied
 * @Param[in] mask The bitmask stream for nulls
 * @Param[in] size The element count of stream
 * @Param[out] results The stream for the result
 * @Param[in] identity The scholar value to be used to replace nulls
 * @Param[in] stream The cuda stream to be used
 *
 * @Returns  If the operation was successful, returns GDF_SUCCESS
 */
/* ----------------------------------------------------------------------------*/
    template <class T>
    inline
        gdf_error copy_and_replace_nulls(
            const T *data, const gdf_valid_type *mask,
            gdf_size_type size, T *results, T identity, hipStream_t stream)
    {
        int blocksize = (size < COPYMASK_BLOCK_SIZE ?
            size : COPYMASK_BLOCK_SIZE);
        int gridsize = (size + COPYMASK_BLOCK_SIZE - 1) /
            COPYMASK_BLOCK_SIZE;

        // launch kernel
        gpu_copy_and_replace_nulls << <gridsize, blocksize, 0, stream >> > (
            data, mask, size, results, identity);

        CUDA_CHECK_LAST();
        return GDF_SUCCESS;
    }

    template <class T>
    struct Scan {
        static
            gdf_error call(const gdf_column *input, gdf_column *output,
                bool inclusive, hipStream_t stream) {
            gdf_error ret;
            auto scan_function = (inclusive ? inclusive_sum : exclusive_sum);
            size_t size = input->size;
            const T* d_input = static_cast<const T*>(input->data);
            T* d_output = static_cast<T*>(output->data);

            // Prepare temp storage
            void *temp_storage = NULL;
            size_t temp_storage_bytes = 0;
            GDF_REQUIRE(GDF_SUCCESS == (ret = scan_function(temp_storage,
                temp_storage_bytes, d_input, d_output, size, stream)), ret);
            RMM_TRY(RMM_ALLOC(&temp_storage, temp_storage_bytes, stream));

            if( nullptr != input->valid ){
                // copy null bitmask
                CUDA_TRY(hipMemcpyAsync(output->valid, input->valid,
                        gdf_num_bitmask_elements(input->size), hipMemcpyDeviceToDevice, stream));
                output->null_count = input->null_count;
            }

            bool const input_has_nulls{ nullptr != input->valid &&
                                        input->null_count > 0 };
            if (input_has_nulls) {
                // allocate temporary column data
                T* temp_input;
                RMM_TRY(RMM_ALLOC(&temp_input, size * sizeof(T), stream));

                // copy d_input data and replace with 0 if mask is null
                copy_and_replace_nulls(
                    static_cast<const T*>(input->data), input->valid,
                    size, temp_input, static_cast<T>(0), stream);

                // Do scan
                ret = scan_function(temp_storage, temp_storage_bytes,
                    temp_input, d_output, size, stream);
                GDF_REQUIRE(GDF_SUCCESS == ret, ret);

                RMM_TRY(RMM_FREE(temp_input, stream));
            }
            else {  // Do scan
                ret = scan_function(temp_storage, temp_storage_bytes,
                    d_input, d_output, size, stream);
                GDF_REQUIRE(GDF_SUCCESS == ret, ret);
            }

            // Cleanup
            RMM_TRY(RMM_FREE(temp_storage, stream));

            return GDF_SUCCESS;
        }

        static
            gdf_error exclusive_sum(void *&temp_storage, size_t &temp_storage_bytes,
                const T *input, T *output, size_t size, hipStream_t stream) {
            hipcub::DeviceScan::ExclusiveSum(temp_storage, temp_storage_bytes,
                input, output, size, stream);
            CUDA_CHECK_LAST();
            return GDF_SUCCESS;
        }

        static
            gdf_error inclusive_sum(void *&temp_storage, size_t &temp_storage_bytes,
                const T *input, T *output, size_t size, hipStream_t stream) {
            hipcub::DeviceScan::InclusiveSum(temp_storage, temp_storage_bytes,
                input, output, size, stream);
            CUDA_CHECK_LAST();
            return GDF_SUCCESS;
        }
    };

    struct PrefixSumDispatcher {
        template <typename T,
            typename std::enable_if_t<std::is_arithmetic<T>::value>* = nullptr>
            gdf_error operator()(const gdf_column *input, gdf_column *output,
                bool inclusive, hipStream_t stream = 0) {
            GDF_REQUIRE(input->size == output->size, GDF_COLUMN_SIZE_MISMATCH);
            GDF_REQUIRE(input->dtype == output->dtype, GDF_DTYPE_MISMATCH);

            if (nullptr == input->valid) {
                GDF_REQUIRE(0 == input->null_count, GDF_VALIDITY_MISSING);
                GDF_REQUIRE(nullptr == output->valid, GDF_VALIDITY_UNSUPPORTED);
            }
            else {
                GDF_REQUIRE(nullptr != input->valid && nullptr != output->valid,
                            GDF_VALIDITY_MISSING);
            }
            return Scan<T>::call(input, output, inclusive, stream);
        }

        template <typename T,
            typename std::enable_if_t<!std::is_arithmetic<T>::value, T>* = nullptr>
            gdf_error operator()(const gdf_column *input, gdf_column *output,
                bool inclusive, hipStream_t stream = 0) {
            return GDF_UNSUPPORTED_DTYPE;
        }
    };

} // end anonymous namespace

gdf_error gdf_prefixsum(const gdf_column *input, gdf_column *output, bool inclusive)
{
    return cudf::type_dispatcher(input->dtype, PrefixSumDispatcher(),
        input, output, inclusive);
}
