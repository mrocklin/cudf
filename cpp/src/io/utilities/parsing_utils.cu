#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
	 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

/**
 * @file parsing_utils.cu Utility functions for parsing plain-text files
 *
 */


#include "parsing_utils.cuh"

#include <hip/hip_runtime.h>

#include <vector>
#include <memory>

#include "rmm/rmm.h"
#include "rmm/thrust_rmm_allocator.h"
#include "utilities/error_utils.hpp"

// When processing the input in chunks, this is the maximum size of each chunk.
// Only one chunk is loaded on the GPU at a time, so this value is chosen to
// be small enough to fit on the GPU in most cases.
constexpr size_t max_chunk_bytes = 256*1024*1024; // 256MB

constexpr int bytes_per_find_thread = 64;

template <typename T>
struct rmm_deleter {
 void operator()(T *ptr) { RMM_FREE(ptr, 0); }
};
template <typename T>
using device_ptr = std::unique_ptr<T, rmm_deleter<T>>;

/**---------------------------------------------------------------------------*
 * @brief Sets the specified element of the array to the passed value
 *---------------------------------------------------------------------------**/
template<class T, class V>
__device__ __forceinline__
void setElement(T* array, gdf_size_type idx, const T& t, const V& v){
	array[idx] = t;
}

/**---------------------------------------------------------------------------*
 * @brief Sets the specified element of the array of pairs using the two passed
 * parameters.
 *---------------------------------------------------------------------------**/
template<class T, class V>
__device__ __forceinline__
void setElement(thrust::pair<T, V>* array, gdf_size_type idx, const T& t, const V& v) {
	array[idx] = {t, v};
}

/**---------------------------------------------------------------------------*
 * @brief Overloads the setElement() functions for void* arrays.
 * Does not do anything, indexing is not allowed with void* arrays.
 *---------------------------------------------------------------------------**/
template<class T, class V>
__device__ __forceinline__
void setElement(void* array, gdf_size_type idx, const T& t, const V& v) {
}

/**---------------------------------------------------------------------------*
 * @brief CUDA kernel that finds all occurrences of a character in the given 
 * character array. If the 'positions' parameter is not void*,
 * positions of all occurrences are stored in the output array.
 * 
 * @param[in] data Pointer to the input character array
 * @param[in] size Number of bytes in the input array
 * @param[in] offset Offset to add to the output positions
 * @param[in] key Character to find in the array
 * @param[in,out] count Pointer to the number of found occurrences
 * @param[out] positions Array containing the output positions
 * 
 * @return void
 *---------------------------------------------------------------------------**/
template<class T>
 __global__ 
 void countAndSetPositions(char *data, uint64_t size, uint64_t offset, const char key, gdf_size_type* count,
	T* positions) {

	// thread IDs range per block, so also need the block id
	const uint64_t tid = threadIdx.x + (blockDim.x * blockIdx.x);
	const uint64_t did = tid * bytes_per_find_thread;
	
	const char *raw = (data + did);

	const long byteToProcess = ((did + bytes_per_find_thread) < size) ?
									bytes_per_find_thread :
									(size - did);

	// Process the data
	for (long i = 0; i < byteToProcess; i++) {
		if (raw[i] == key) {
			const auto idx = atomicAdd(count, (gdf_size_type)1);
			setElement(positions, idx, did + offset + i, key);
		}
	}
}

/**---------------------------------------------------------------------------*
 * @brief Searches the input character array for each of characters in a set.
 * Sums up the number of occurrences. If the 'positions' parameter is not void*,
 * positions of all occurrences are stored in the output device array.
 * 
 * Does not load the entire file into the GPU memory at any time, so it can 
 * be used to parse large files. Output array needs to be preallocated.
 * 
 * @param[in] h_data Pointer to the input character array
 * @param[in] h_size Number of bytes in the input array
 * @param[in] keys Vector containing the keys to count in the buffer
 * @param[in] result_offset Offset to add to the output positions
 * @param[out] positions Array containing the output positions
 * 
 * @return gdf_size_type total number of occurrences
 *---------------------------------------------------------------------------**/
template<class T>
gdf_size_type findAllFromSet(const char *h_data, size_t h_size, const std::vector<char>& keys, uint64_t result_offset,
	T *positions) {

	char* d_chunk = nullptr;
	RMM_TRY(RMM_ALLOC (&d_chunk, min(max_chunk_bytes, h_size), 0));
	device_ptr<char> chunk_deleter(d_chunk);

	gdf_size_type*	d_count;
	RMM_TRY(RMM_ALLOC((void**)&d_count, sizeof(gdf_size_type), 0) );
	device_ptr<gdf_size_type> count_deleter(d_count);
	CUDA_TRY(hipMemsetAsync(d_count, 0ull, sizeof(gdf_size_type)));

	int blockSize;		// suggested thread count to use
	int minGridSize;	// minimum block count required
	CUDA_TRY(hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, countAndSetPositions<T>) );

	const size_t chunk_count = (h_size + max_chunk_bytes - 1) / max_chunk_bytes;
	for (size_t ci = 0; ci < chunk_count; ++ci) {	
		const auto chunk_offset = ci * max_chunk_bytes;	
		const auto h_chunk = h_data + chunk_offset;
		const auto chunk_bytes = std::min((size_t)(h_size - ci * max_chunk_bytes), max_chunk_bytes);
		const auto chunk_bits = (chunk_bytes + bytes_per_find_thread - 1) / bytes_per_find_thread;
		const int gridSize = (chunk_bits + blockSize - 1) / blockSize;

		// Copy chunk to device
		CUDA_TRY(hipMemcpyAsync(d_chunk, h_chunk, chunk_bytes, hipMemcpyDefault));

		for (char key: keys) {
			countAndSetPositions<T> <<< gridSize, blockSize >>> (
				d_chunk, chunk_bytes, chunk_offset + result_offset, key,
				d_count, positions);
		}
	}

	gdf_size_type h_count = 0;
	CUDA_TRY(hipMemcpy(&h_count, d_count, sizeof(gdf_size_type), hipMemcpyDefault));
	return h_count;
}

/**---------------------------------------------------------------------------*
 * @brief Searches the input character array for each of characters in a set
 * and sums up the number of occurrences.
 *
 * Does not load the entire buffer into the GPU memory at any time, so it can 
 * be used with buffers of any size.
 *
 * @param[in] h_data Pointer to the data in host memory
 * @param[in] h_size Size of the input data, in bytes
 * @param[in] keys Vector containing the keys to count in the buffer
 *
 * @return gdf_size_type total number of occurrences
 *---------------------------------------------------------------------------**/
gdf_size_type countAllFromSet(const char *h_data, size_t h_size, const std::vector<char>& keys) {
	return findAllFromSet<void>(h_data, h_size, keys, 0, nullptr);
 }

template gdf_size_type findAllFromSet<uint64_t>(const char *h_data, size_t h_size, const std::vector<char>& keys, uint64_t result_offset,
	uint64_t *positions);

template gdf_size_type findAllFromSet<thrust::pair<uint64_t,char>>(const char *h_data, size_t h_size, const std::vector<char>& keys, uint64_t result_offset,
	thrust::pair<uint64_t,char> *positions);
